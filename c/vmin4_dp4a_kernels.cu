
#include <hip/hip_runtime.h>
__global__ void vmin4(
	int32_t &d,
	unsigned const &A,
	unsigned const &B,
	int32_t const &c){
	#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 300))
		asm volatile("vmin4.s32.s32.s32.add %0, %1, %2, %3;"
                 : "=r"(d)
                 : "r"(A), "r"(B), "r"(c));
	#endif
}

__global__ void vmin4_intrinsic(
	int32_t &d,
	unsigned const &A,
	unsigned const &B,
	int32_t const &c){
	#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 300))
		int8_t t[4];
		unsigned &T = reinterpret_cast<unsigned &>(t);
		T = __vmins4(A, B);
		d += t[0] + t[1] + t[2] + t[3];
	#endif
}

__global__ void vmin4_intrinsic_dp4a(
	int32_t &d,
	unsigned const &A,
	unsigned const &B,
	int32_t const &c){
	#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610))
		int32_t T = __vmins4(A, B);
		d = __dp4a(T, 0x01010101, c);
	#endif
}

__global__ void dp4a(
    int32_t &d,
    unsigned const &A,
    unsigned const &B,
    int32_t const &c){
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610))
        asm volatile("dp4a.s32.s32 %0, %1, %2, %3;"
                 : "=r"(d)
                 : "r"(A), "r"(B), "r"(c));
#endif
}

__global__ void loop_unroll(
    int32_t &d,
    int8_t const a[],
    int8_t const b[],
    int32_t const &c){
    d = c;

    #pragma unroll
    for (int k = 0; k < 4; ++k) {
      d += a[k] * b[k];
    }
}


